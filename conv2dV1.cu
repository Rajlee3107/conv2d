#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <fstream>
#include <iomanip>

void readInput(const char* fileName, float*& h_input, int& H, int& W) {
    std::ifstream inFile(fileName);
    if (!inFile) {
        std::cerr << "Error: Could not open Input file " << fileName << std::endl;
        exit(EXIT_FAILURE);
    }

    inFile >> H >> W; // Using the extraction operator
    int size = H * W * sizeof(float);
    // Allocate the host input matrix
    h_input = (float *)malloc(size);

    for(int i = 0; i < H * W; i++) {
      if(!(inFile >> h_input[i])) {
	std::cerr << "Error: Invalid file format. Expected " << H * W << " elements." << std::endl;
        exit(EXIT_FAILURE); 
      }
    }
  }

void readFilter(const char* fileName, float*& h_filter, int& R) {
  std::ifstream inFile(fileName);
  if(!inFile) {
    std::cerr << "Error: Could not open Filter file " << fileName << std::endl;
    exit(EXIT_FAILURE);
  }

  inFile >> R;
  int size = R * R * sizeof(float);
  // Allocate the host filter matrix
  h_filter = (float *)malloc(size);

  for(int i = 0; i < R * R; i++) {
    if(!(inFile >> h_filter[i])) {
      std::cerr << "Error: Invalid file format. Expected " << R * R << " elements." << std::endl;
      exit(EXIT_FAILURE); 
    }
  }
}

void writeOutput(float* h_output, int H, int W) {
  
  for(int i = 0; i < H; i++) {
    for(int j = 0; j < W; j++) {
      std::cout << std::fixed << std::setprecision(3) << h_output[i * W + j] << std::endl;
    }
  }
}


//------- The cuda kernel -------//

// Both the input and the filter will have float values in them.
__global__ void conv2d_kernel(float* input, float* filter, float* output, int H, int W, int R) { 
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < H && col < W) {
    float sum = 0.0f;
    int pad = R/2;
    for(int i = -pad; i <= pad; i++) {
      for(int j = -pad; j <= pad; j++) {
        int r = row + i;
        int c = col + j;
        if(r >= 0 && r < H && c >= 0 && c < H) {
          int filter_row = i +pad;
          int filter_col = j + pad;
          sum += input[r * W + c] * filter[filter_row * R + filter_col];
        }
      }
    }
    output[row * W + col] = sum;
  }
}


int main(int argc, char *argv[]) {

  if(argc < 3) {
    std::cerr << "ERROR: number of arguments < 3" << std::endl;
    return -1;
  }

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // ---- Read the inputs from command line ---- //
  int H, W, R;
  float *h_input, *h_filter;
  readInput(argv[1], h_input, H, W);
  readFilter(argv[2], h_filter, R);
  printf("[Vector multiplication of %0d x %0d by %0d x %0d matrices]\n", H, W, R, R);

  // ---- Allocate/move data using cudaMalloc and cudaMemCpy ---- //
  // Verify that allocations succeeded
  if (h_input == NULL || h_filter == NULL)
  {
    fprintf(stderr, "Failed to allocate host input and filter matrices!\n");
    exit(EXIT_FAILURE);
  }

  // Allocate the device input matrix
    float *d_input = NULL;
    err = hipMalloc((void **)&d_input, H * W * sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_filter = NULL;
    err = hipMalloc((void **)&d_filter, R * R * sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix filter (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_output = NULL;
    err = hipMalloc((void **)&d_output, H * W * sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input matrices input and filter in host memory to the device input and filter matrices in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_input, h_input, H * W * sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix input from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_filter, h_filter, R * R * sizeof(float), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix filter from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  // ---- Launch the kernel ---- //
  // Define grid and block dimensions
  dim3 blockDim(16, 16);
  dim3 gridDim((W + 15) / 16, (H + 15) / 16);

  printf("CUDA kernel launch with %d blocks of %d threads\n", gridDim, blockDim);
  conv2d_kernel<<<gridDim, blockDim>>>(d_input, d_filter, d_output, H, W, R);
  err = hipGetLastError();

  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch 2D convolution kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  // ----- Copy results back to host ----- //
  // Allocate the host output matrix
  float *h_output = (float *)malloc(H * W * sizeof(float));

  // Verify that allocations succeeded
  if (h_output == NULL)
  {
    fprintf(stderr, "Failed to allocate host output matrix!\n");
    exit(EXIT_FAILURE);
  }

  printf("Copy output matrix from the CUDA device to the host memory\n");
  err = hipMemcpy(h_output, d_output, H * W * sizeof(float), hipMemcpyDeviceToHost);

  if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output matrix from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  // Print the output
  writeOutput(h_output, H, W);
  printf("Test Completed\n");

  // ---- Clean up the memory ----- //
  err = hipFree(d_input);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device input matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_filter);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device filter matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device output matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_input);
    free(h_filter);
    free(h_output);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");


  return 0;
}
